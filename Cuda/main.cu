#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define INPUT_FILE_ENDING "dat"
#define MAX_CIRCLES_DEFAULT 5
#define MAX_CIRCLES_MAX 10
#define CLUSTER_SIZE_DEFAULT 10
#define CLUSTER_SIZE_MAX 100
#define ITERATIONS_MAX_DEFAULT 100
#define OVERFIT_PENALTY 0.001
#define CIRCLE_CENTER_LOW  -1.0
#define CIRCLE_CENTER_HIGH  1.0
#define CIRCLE_RADIUS_LOW   0.1
#define CIRCLE_RADIUS_HIGH  1.0

typedef struct Point {
	double x;
	double y;
} POINT; 

typedef struct Circle {
	POINT center;
	double radius;
} CIRCLE;

typedef struct CircleFitData {
	int N;
	double Mx, My;
	double Suu, Suuu;
	double Svv, Svvv;
	double Suv, Suvv, Svuu;
} CIRCLE_FIT_DATA;

__device__ const CIRCLE_FIT_DATA CFD_0 = {0, 0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0};

void usage(const char* program_name);
void file_error(const char *error_msg, FILE *fd);
void cudaCheckErrors(hipError_t error);
int outputCircles(FILE* stream, CIRCLE* circles, int* returnedCircles, int eventsNum, int maxCircles, int eventLow, double* LADs);

//Implementation of CUDA's atomicAdd() for double data, since it is officially implemented by CUDA 
//Still, the code is given as a suggestion in the documentation of CUDA
__device__ double atomicAdd(double* address, double val); 

/****************************** CUDA Kernels ******************************/

//One thread per point, one block per speculated number of circles per column, one column per event in a cluster
__global__ void fitCirclesKernel(POINT *d_points, CIRCLE *d_circles, int* d_pointsCumSums, double *d_LDSs, int maxCircles, hiprandState_t* statesRNG){
//Shared Variables
	extern __shared__ int sharedMem[];	
	CIRCLE *C = (CIRCLE*)sharedMem;
	CIRCLE_FIT_DATA *CFD = (CIRCLE_FIT_DATA*)&C[maxCircles];
	int *somethingChanged = (int*)&CFD[maxCircles];
	float *LDS = (float*)&somethingChanged[1];
	
//Local Variables
	int pointsNum = d_pointsCumSums[0];
	int circlesNum = blockIdx.x+1;
	int pointID = threadIdx.x;	
	POINT thePoint;
	
	double distance;
	double minDist;
	int belongsToCircle, newClosestCircle;
	double u, v;
	int i, iterations;
	int iterationsMax = ITERATIONS_MAX_DEFAULT;
	
//Identify the correct starting point of the memory, for each event
	if(blockIdx.y != 0){
		d_points += d_pointsCumSums[blockIdx.y-1];
		pointsNum = d_pointsCumSums[blockIdx.y] - d_pointsCumSums[blockIdx.y-1]; 
	}	
	d_circles += (blockIdx.y*maxCircles + blockIdx.x) * maxCircles;		
	d_LDSs += blockIdx.y*maxCircles + blockIdx.x;
	thePoint = d_points[pointID];
	
//Proceed with the Kernel Execution only for the appropriate amount of threads	
	if(pointID < pointsNum){		
	
	/***** KCC - Create the initial Circles *****/	
	
	//Random Circle Initialization		
		if(pointID < circlesNum){
			int idRNG, min, max;
			idRNG = blockIdx.y*maxCircles*maxCircles + blockIdx.x*maxCircles + pointID;	
			
			min = CIRCLE_CENTER_LOW;
			max = CIRCLE_CENTER_HIGH;
			C[pointID].center.x = hiprand_uniform_double(&statesRNG[idRNG]) * (max - min) + (min);
			C[pointID].center.y = hiprand_uniform_double(&statesRNG[idRNG]) * (max - min) + (min);
		
			min = CIRCLE_RADIUS_LOW;
			max = CIRCLE_RADIUS_HIGH;
			C[pointID].radius = hiprand_uniform_double(&statesRNG[idRNG]) * (max - min) + (min);			
		}
		__syncthreads();
		
		belongsToCircle = -1;
		iterations = 0;
		do{
			*somethingChanged = 0;
	/***** KCC - ITERATIVE STEP 1: Assing every Point to it's closet Circle *****/
		
		//Calculate distance from every Circle, select min	
			minDist = 100000.0;
			newClosestCircle = -1;
			for(i=0; i<circlesNum; i++){
				distance = sqrt( pow((thePoint.x - C[i].center.x), 2) + pow((thePoint.y - C[i].center.y), 2) );
				distance = fabs( distance - C[i].radius );
				if(distance < minDist){
					minDist = distance;
					newClosestCircle = i;
				}	
			}	
			if(newClosestCircle != belongsToCircle){
				belongsToCircle = newClosestCircle;
				*somethingChanged = 1;
			}
		
	/***** KCC - ITERATIVE STEP 2: Fit every Circle to it's Points using least squares *****/	
		
		//Set every CFD variable of every Circle to 0
			if(pointID < circlesNum){
				CFD[pointID] = CFD_0;		
			}
			__syncthreads();
			
		//Calculate the means of the Points coords, in every Circle
			atomicAdd(&CFD[belongsToCircle].Mx, thePoint.x);
			atomicAdd(&CFD[belongsToCircle].My, thePoint.y);
			atomicAdd(&CFD[belongsToCircle].N, 1);	
			__syncthreads();
			
			if(pointID < circlesNum){
				CFD[pointID].Mx /= CFD[pointID].N;
				CFD[pointID].My /= CFD[pointID].N;
			}
			__syncthreads();
			u = thePoint.x - CFD[belongsToCircle].Mx;
			v = thePoint.y - CFD[belongsToCircle].My;
		
		//Create the 2x2 linear system, used to find the best fitted Circle (using least squares method)
			atomicAdd(&CFD[belongsToCircle].Suu,  u*u);
			atomicAdd(&CFD[belongsToCircle].Suuu, u*u*u);
			atomicAdd(&CFD[belongsToCircle].Svv,  v*v);
			atomicAdd(&CFD[belongsToCircle].Svvv, v*v*v);
			atomicAdd(&CFD[belongsToCircle].Suv,  u*v);
			atomicAdd(&CFD[belongsToCircle].Suvv, u*v*v);
			atomicAdd(&CFD[belongsToCircle].Svuu, v*u*u);
		
		//Solve the the 2x2 linear system, by using the closed type for A_inv
			__syncthreads();	
			if(pointID < circlesNum){		
				double det = CFD[pointID].Suu*CFD[pointID].Svv - CFD[pointID].Suv*CFD[pointID].Suv;	
				
				double b[2];
				b[0] = (CFD[pointID].Suuu + CFD[pointID].Suvv)/2;
				b[1] = (CFD[pointID].Svvv + CFD[pointID].Svuu)/2;
					
				double u_c = ( CFD[pointID].Svv*b[0] - CFD[pointID].Suv*b[1])/det;
				double v_c = (-CFD[pointID].Suv*b[0] + CFD[pointID].Suu*b[1])/det;		
				double r = u_c*u_c + v_c*v_c + (CFD[pointID].Suu + CFD[pointID].Svv)/CFD[pointID].N;
				
				C[pointID].center.x = u_c + CFD[pointID].Mx;
				C[pointID].center.y = v_c + CFD[pointID].My;	
				C[pointID].radius = sqrt(r);
			}
			__syncthreads();
		
	//Iterate until either max number of iterations is reached, 
	//or no change happened in assignment of Points to Circles from the previous iteration
			iterations++;
		}while( (iterations < iterationsMax) && (*somethingChanged == 1) );			
		
	/***** Calculate Each Solution's LDS *****/
		
	//Each thread-point will calculate it's minimum distance from all circles
		if(pointID == 0){			
			*LDS = OVERFIT_PENALTY*circlesNum*circlesNum; // Add the overfitting penalty
		}
		__syncthreads();
		
		minDist = 100000.0;
		for(i=0; i<circlesNum; i++){
			distance = sqrt( pow((thePoint.x - C[i].center.x), 2) + pow((thePoint.y - C[i].center.y), 2) );
			distance = fabs( distance - C[i].radius );
			if(distance < minDist){
				minDist = distance;			
			}	
		}	
		atomicAdd(&LDS[0], minDist);		
		__syncthreads();	
		
		if(pointID == 0){
			*d_LDSs = *LDS;
		}	
		__syncthreads();
	
	//Write-back the computed circles			
		if(pointID < circlesNum){
			d_circles[pointID].center.x  = C[pointID].center.x;
			d_circles[pointID].center.y  = C[pointID].center.y;
			d_circles[pointID].radius  = C[pointID].radius;
		}			
		
	}		
}

//One thread per point, one block per event, to find the minimum LDS
//and return the corresponding solution and it's computed LAD
__global__ void computeSolutionKernel(POINT *d_points, CIRCLE *d_circles, int* d_pointsCumSums, double *d_LDSs, double *d_LADs, int* d_returnedCirclesNum, int maxCircles){
//Shared Variables
	extern __shared__ int sharedMem[];	
	float *LAD = (float*)sharedMem;

//Local Variables
	int pointsNum;
	int pointID, eventID;	
	POINT thePoint;
	int bestSolution, returnedCirclesNum;
	double minLDS, deviation, LADp;
	int i, t;
	CIRCLE *circles_input;
	double cX, cY, cR;
	
//Identify the correct starting point of the memory, for each event	
	pointID = threadIdx.x;
	eventID = blockIdx.x;	
	pointsNum = d_pointsCumSums[0];
	if(eventID != 0){
		d_points += d_pointsCumSums[eventID-1];
		pointsNum = d_pointsCumSums[eventID] - d_pointsCumSums[eventID-1]; 
	}		
	d_circles += eventID * maxCircles * maxCircles;		
	d_LDSs += eventID * maxCircles;
	d_returnedCirclesNum += eventID;
	d_LADs += eventID;
	thePoint = d_points[pointID];

//Find the smallest LDS and the returnedCirclesNum of that solution	
	if(pointID == 0){
		bestSolution = 0;
		minLDS = d_LDSs[0];
		for(i=0; i<maxCircles; i++){
			if(d_LDSs[i] < minLDS){
				minLDS = d_LDSs[i];
				bestSolution = i;
			}		
		}
		returnedCirclesNum = bestSolution+1;
		circles_input = d_circles + bestSolution*maxCircles;
	}
	
//Write back the best solution
	if(pointID == 0){
		if(bestSolution != 0){ //If (bestSolution == 0) your don't need to move the results
			t=0;
			for(i=0; i<returnedCirclesNum; i++){
				cX = circles_input[i].center.x;
				cY = circles_input[i].center.y;
				cR = circles_input[i].radius;
				
				if( isnan(cX) == 0 && isnan(cY) == 0 && isnan(cR) == 0){
					d_circles[t].center.x = cX;
					d_circles[t].center.y = cY;
					d_circles[t].radius = cR;
					t++;	
				}
			}
		}			
		*d_returnedCirclesNum = t;
	}
	__syncthreads();
	returnedCirclesNum = *d_returnedCirclesNum;
	
//Calculate and write the LAD metric	
	if(pointID < pointsNum){
		if(pointID == 0){
			*LAD = 0;
		}
		__syncthreads();
		
		LADp = 100000.0;
		for(i=0; i<returnedCirclesNum; i++){
			cX = d_circles[i].center.x;
			cY = d_circles[i].center.y;
			cR = d_circles[i].radius;
			
			deviation = pow((thePoint.x - cX), 2) + pow((thePoint.y - cY), 2) - pow(cR, 2);
			deviation = pow(deviation, 2);
			if(deviation < LADp){
				LADp = deviation;	
			}
		}
		atomicAdd(&LAD[0], LADp);		
		__syncthreads();	
		
		if(pointID == 0){
			*d_LADs = *LAD;
		}
	}				
}
	
__global__ void initRNG(int seed, hiprandState_t* statesRNG){
	hiprand_init(seed, blockIdx.x, 0, &statesRNG[blockIdx.x]);
}

/******************************    Main    ******************************/
int main(int argc, char** argv){
//CPU (host) variables	
	char *ext;
	FILE *fd = NULL;
	int i, event, clusterPart, begin;
	int sharedMemorySize;
	hiprandState_t* statesRNG;
	struct timeval t1, t2;
	
	int eventsNum, pointsNum;
	int pointsNumMax, pointsArraySize;
	double pointX, pointY;
	double errorPercentage = 0.0;
	int maxCircles;
	int clusterSize;
	
	POINT *points;
	CIRCLE *circles;
	double *LADs;
	int *returnedCirclesNum;	
	int *pointsCumSums;
	
//Device variables
	POINT *d_points;
	CIRCLE *d_circles;
	double *d_LADs;
	int *d_returnedCirclesNum;
	int *d_pointsCumSums;
	double *d_LDSs;

//Commmand line arguments Validation
	if(argc < 2){
		printf("Input was file not specified\n");
		usage(argv[0]);
		exit(1);
	}
	ext = strrchr(argv[1], '.');
	if(ext==NULL || strcmp(ext+1, INPUT_FILE_ENDING) != 0){
		printf("Unrecognized input file ending \n");
		usage(argv[0]);
		exit(1);
	}
	
	maxCircles = MAX_CIRCLES_DEFAULT;
	if(argc > 2){
		char *temp;			
		maxCircles = (int)strtol(argv[2], &temp, 0);		
		if(*temp != '\0' || maxCircles<1){
			printf("Second argument [%s] is not an positive integer number \n", argv[2]);
			usage(argv[0]);
			exit(1);
		}
		if(maxCircles > MAX_CIRCLES_MAX){
			printf("Value [%d] for max_circles_size is too big \n", maxCircles);
			printf("Setting to the max accepted value: %d\n", MAX_CIRCLES_MAX);
			maxCircles = MAX_CIRCLES_MAX;
		}
	}		
	clusterSize = CLUSTER_SIZE_DEFAULT;
	if(argc > 3){
		char *temp;			
		clusterSize = (int)strtol(argv[3], &temp, 0);		
		if(*temp != '\0' || clusterSize<1){
			printf("Third argument [%s] is not an positive integer number \n", argv[3]);
			usage(argv[0]);
			exit(1);
		}
		if(clusterSize > CLUSTER_SIZE_MAX){
			printf("Value [%d] for cluster_size is too big \n", clusterSize);
			printf("Setting to the max accepted value: %d\n", CLUSTER_SIZE_MAX);
			clusterSize = CLUSTER_SIZE_MAX;
		}
	}

	gettimeofday(&t1, NULL); //Begin Timing
	
//Input file (.dat) Parsing
	if((fd = fopen(argv[1], "r")) == NULL){
		file_error("Error while opening specified input file", fd);
		exit(1);
	}
	
	eventsNum = pointsNum = 0;
	pointX = pointY = 0.0;
	
	fscanf(fd, "%d", &eventsNum);
	if(ferror(fd) != 0){
		file_error("Error while reading from the input file", fd);
		exit(1);
	}	
	
//Allocate Host - Device Memory with static size
	circles = (CIRCLE *)malloc(clusterSize*maxCircles*maxCircles*sizeof(CIRCLE));
	LADs = (double *)malloc(clusterSize*sizeof(double));
	returnedCirclesNum = (int *)malloc(clusterSize*sizeof(int));
	pointsCumSums = (int *)malloc(clusterSize*sizeof(int));
	
	cudaCheckErrors(hipMalloc((void**) &d_circles, clusterSize*maxCircles*maxCircles*sizeof(CIRCLE)));		
	cudaCheckErrors(hipMalloc((void**) &d_LADs, clusterSize*sizeof(double)));	
	cudaCheckErrors(hipMalloc((void**) &d_returnedCirclesNum, clusterSize*sizeof(int)));
	cudaCheckErrors(hipMalloc((void**) &d_pointsCumSums, clusterSize*sizeof(int)));
	cudaCheckErrors(hipMalloc((void**) &d_LDSs, clusterSize*maxCircles*sizeof(double)));
	
//Initialize the RNG states for every thread that will randomly initialize a circle
	int N = maxCircles*maxCircles*clusterSize;
	cudaCheckErrors(hipMalloc((void**) &statesRNG, N*sizeof(hiprandState_t)));
	initRNG<<<1, N>>>(time(0), statesRNG);

//Start Loop
	
	clusterPart = 0;
	for(event=0; event<eventsNum; event++){	
	//Read an event
		fscanf(fd, "%d", &pointsNum);
		if(ferror(fd) != 0){
			file_error("Error while reading from the input file", fd);
			exit(1);
		}
		
	//Allocate-Initialize Host Memory with dynamic size		
		pointsCumSums[clusterPart] = pointsNum;
		if(clusterPart == 0){ //First event of the cluster
			pointsCumSums[clusterPart] = pointsNum;
			begin = 0;
			pointsNumMax = pointsNum;
			points = (POINT *)malloc(pointsNum*sizeof(POINT));
		}else{		
			pointsCumSums[clusterPart] = pointsCumSums[clusterPart-1] + pointsNum;
			begin = pointsCumSums[clusterPart-1];
			if(pointsNum > pointsNumMax){
				pointsNumMax = pointsNum;
			}
			points = (POINT *)realloc(points, pointsCumSums[clusterPart]*sizeof(POINT));
		}				
		for(i=begin; i<pointsCumSums[clusterPart]; i++){
			fscanf(fd, "%lf", &pointX);
			if(ferror(fd) != 0){
				file_error("Error while reading from the input file", fd);
				exit(1);
			}
			fscanf(fd, "%lf", &pointY);
			if(ferror(fd) != 0){
				file_error("Error while reading from the input file", fd);
				exit(1);
			}
			points[i].x = pointX;
			points[i].y = pointY;
		}
		sharedMemorySize = maxCircles*( sizeof(CIRCLE) + sizeof(CIRCLE_FIT_DATA) ) + sizeof(int) + sizeof(float);
		
		clusterPart++;
	// The cluster is full; Proceed with the Kernel call	
		if( (clusterPart == clusterSize) || (event == eventsNum-1) ){
			
		//Allocate-Initialize Device Memory that has dynamic size
			pointsArraySize = pointsCumSums[clusterPart-1]*sizeof(POINT);
			cudaCheckErrors(hipMalloc((void**) &d_points, pointsArraySize));	
			cudaCheckErrors(hipMemcpy(d_points, points, pointsArraySize, hipMemcpyHostToDevice));			
			cudaCheckErrors(hipMemcpy(d_pointsCumSums, pointsCumSums, clusterSize*sizeof(int), hipMemcpyHostToDevice));	
			
		//Call the CUDA Kernel
			const dim3 blockSize(pointsNumMax, 1, 1);
			const dim3 gridSize1(maxCircles, clusterPart, 1);
			const dim3 gridSize2(clusterPart, 1, 1);
			
			fitCirclesKernel<<<gridSize1, blockSize, sharedMemorySize>>>(d_points, d_circles, d_pointsCumSums, d_LDSs, maxCircles, statesRNG);	
			cudaCheckErrors(hipDeviceSynchronize()); // DEBUG Is it needed?
			computeSolutionKernel<<<gridSize2, blockSize, sizeof(float)>>>(d_points, d_circles, d_pointsCumSums, d_LDSs, d_LADs, d_returnedCirclesNum, maxCircles);
			
		//Display Results
			cudaCheckErrors(hipMemcpy(circles, d_circles, clusterSize*maxCircles*maxCircles*sizeof(CIRCLE), hipMemcpyDeviceToHost));
			cudaCheckErrors(hipMemcpy(LADs, d_LADs, clusterSize*sizeof(double), hipMemcpyDeviceToHost));
			cudaCheckErrors(hipMemcpy(returnedCirclesNum, d_returnedCirclesNum, clusterSize*sizeof(int), hipMemcpyDeviceToHost));
			
			errorPercentage += outputCircles(NULL, circles, returnedCirclesNum, clusterPart, maxCircles, event-clusterPart+1, LADs);
			
		//Deallocate Host - Device Memory with dynamic size
			free(points);
			cudaCheckErrors(hipFree(d_points)); 			
			
			clusterPart = 0;
		}
	}
	gettimeofday(&t2, NULL);
	fclose(fd);	
	
//Deallocate Host - Device Memory with static size
	free(circles);
	free(LADs);
	free(returnedCirclesNum);
	free(pointsCumSums);	
	
	cudaCheckErrors(hipFree(d_circles));
	cudaCheckErrors(hipFree(d_LADs));
	cudaCheckErrors(hipFree(d_returnedCirclesNum));
	cudaCheckErrors(hipFree(d_pointsCumSums));	
	cudaCheckErrors(hipFree(d_LDSs));
	cudaCheckErrors(hipFree(statesRNG));

	printf("------------------------------\n");
	printf("Input file: %s - Total events: %d\n", argv[1], eventsNum);
	printf("Parallel procedure's execution time: %.3lf ms\n", ((t2.tv_sec-t1.tv_sec)*1000000.0+t2.tv_usec-t1.tv_usec)/1000.0);
	printf("Overall percentage of non-accurate solutions: %.3lf%%\n\n", errorPercentage*100/eventsNum);
	return 0;
}

void usage(const char* program_name){
	printf("Usage: %s <input_file.%s> [max_number_of_circles] [cluster_size]\n", program_name, INPUT_FILE_ENDING);
}

void file_error(const char *error_msg, FILE *fd){
	perror(error_msg);
	if(fd != NULL){
		fclose(fd);
	}	
}

void cudaCheckErrors(hipError_t error){
	if(error != hipSuccess){
		printf("Cuda Error: %s \n", hipGetErrorString(error));
		exit(1);
	}
}

__device__ double atomicAdd(double* address, double val){
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, 
		__double_as_longlong(val + 
		__longlong_as_double(assumed)));
	} while (assumed != old);
	
	return __longlong_as_double(old);
}

int outputCircles(FILE* stream, CIRCLE* circles, int* returnedCircles, int eventsNum, int maxCircles, int eventLow, double* LADs){
	int ev, i, p;
	char cx, cy;
	int errors = 0;
	int noPrint = 0;
	
	if(stream == NULL){
		noPrint = 1;
	}
	
	for(ev=0; ev<eventsNum; ev++){
		(noPrint) || fprintf(stream, "------------------------------\n");
		(noPrint) || fprintf(stream, "Event: %d - Circles found: %d\n", eventLow+ev+1, returnedCircles[ev]);
		for(i=0; i<returnedCircles[ev]; i++){
			p = ev*maxCircles*maxCircles + i;
			cx = (circles[p].center.x < 0) ? '\0' : ' ' ;
			cy = (circles[p].center.y < 0) ? '\0' : ' ' ;
			(noPrint) || fprintf(stream, " CenterX: %c%.10lf | CenterY: %c%.10lf | Radius: %.10lf \n", cx, circles[p].center.x, cy, circles[p].center.y, circles[p].radius);
		}		
		if( LADs[ev] <= pow(10,-10) ){
			(noPrint) || fprintf(stream, "Accurate Solution. LAD Metric < 10^(-10) \n");
		}else{
			(noPrint) || fprintf(stream, "Non-Accurate Solution. LAD Metric: %.10lf > 10^(-10) \n", LADs[ev]);
			errors++;
		}		
	}		
	
	return errors;
}